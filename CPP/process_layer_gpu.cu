#include "hip/hip_runtime.h"
﻿// Cuda base


#include "hip/hip_runtime.h"

#include ""
#define __HIPCC__
#include <hip/hip_runtime_api.h>
#include <cuda_d3d11_interop.h>
#include <hip/device_functions.h>
#include <plog/Log.h>

#include "process_layer_gpu.h"

#include <psapi.h>


#define DARK_MODE_WARP_SIZE 32
#define GLASS_MODE_WARP_SIZE 25
#define GLASS_MODE_WARP_SIZE_SQRT 5
#define GLASS_MODE_COLOR_DIV 11


#define CudaCheckError(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(const hipError_t code, const char* file, const int line, const bool abort = false)
{
	if (code != hipSuccess)
	{
		printf("gpu_assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

namespace process_layer_gpu
{
	// ID3D11DeviceContext* d3d_context = nullptr; // TODO: Maybe remove this

	int x_size, y_size; // x and y size of the texture
	int x_end, y_end; // x and y size of the frame inside the texture
	bool* d_image_area_data = nullptr;
	unsigned char* d_pixels = nullptr;
	unsigned char* d_cached_pixels = nullptr;
	hipArray* cu_array = nullptr;

	bool is_enable_cached_buffer = false;
	hipGraphicsResource* cuda_resource = nullptr;

	bool* d_is_new_pixels = nullptr;
	int* d_bright_pixels_count = nullptr;

	ID3D11DeviceContext* d3d_context{nullptr};

	namespace glass_effect
	{
		bool is_enabled = false;

		unsigned char* pixels_reduced = nullptr;
		unsigned char* d_pixels_reduced = nullptr;

		int x_size_reduced;
		int y_size_reduced;
		int xy_size_reduced;

		double background_level;
		double dark_background;
		double images_level;
		double shapes_level;


		void enable(const double glass_background, const bool glass_dark_background, const double glass_images,
		            const double glass_shapes)
		{
			glass_effect::background_level = glass_background;
			glass_effect::dark_background = glass_dark_background;
			glass_effect::images_level = glass_images;
			glass_effect::shapes_level = glass_shapes;

			is_enabled = true;
		}

		void disable()
		{
			is_enabled = false;
		}

		void dispose()
		{
			// Free GPU memory
			if (d_pixels_reduced)
			{
				hipFree(d_pixels_reduced);
				d_pixels_reduced = nullptr;
			}

			// Free CPU memory
			if (pixels_reduced)
			{
				free(pixels_reduced);
				pixels_reduced = nullptr;
			}
		}

		bool init()
		{
			auto on_error = [&](const char* error_message)
			{
				PLOGE << "load_frame failed: " << error_message;

				dispose();
				return nullptr;
			};

			dispose();

			// Init variables
			x_size_reduced = x_end / GLASS_MODE_WARP_SIZE_SQRT + 1;
			y_size_reduced = y_end / GLASS_MODE_WARP_SIZE_SQRT + 1;
			xy_size_reduced = x_size_reduced * y_size_reduced;

			// Allocate memory

			// Allocate memory in GPU
			const auto result = hipMalloc(&d_pixels_reduced, sizeof(unsigned char) * xy_size_reduced);

			if (result != hipSuccess)
				return on_error("Failed to malloc d_pixels_reduced on GPU");

			// Allocate memory in CPU
			pixels_reduced = static_cast<unsigned char*>(malloc(sizeof(unsigned char) * xy_size_reduced));
			if (pixels_reduced == nullptr) return on_error("Failed to malloc pixels_reduced on CPU");

			return true;
		}


		__global__ void kernel_perform_images_opacity(unsigned char* pixels, int x_size, int y_size,
		                                              const int xy_size,
		                                              const bool* image_area,
		                                              const float images_level)
		{
			const auto thread_4_point = blockIdx.x * blockDim.x + threadIdx.x;
			const auto thread_point = thread_4_point >> 2;

			if (thread_point >= xy_size) return;
			if (!image_area[thread_point]) return;

			pixels[thread_4_point] *= images_level;
		}


		__global__ void kernel_build_reduced_pixels(const unsigned char* pixels, const int x_size,
		                                            const int xy_size, unsigned char* pixels_reduced,
		                                            const int x_reduced)
		{
			// Calculate the points on the screen based on thread and block id
			const auto block_x = (blockIdx.x * GLASS_MODE_WARP_SIZE_SQRT) % x_size;
			const auto block_y = ((blockIdx.x * GLASS_MODE_WARP_SIZE_SQRT) / x_size) * GLASS_MODE_WARP_SIZE_SQRT;
			const auto thread_x = threadIdx.x % GLASS_MODE_WARP_SIZE_SQRT;
			const auto thread_y = threadIdx.x / GLASS_MODE_WARP_SIZE_SQRT;


			const int block_point = x_size * block_y + block_x;
			int thread_point = block_point + thread_y * x_size + thread_x;


			thread_point <<= 2;
			// End


			__shared__ int colors[GLASS_MODE_WARP_SIZE];

			// Mark common colors of these pixels

			colors[threadIdx.x] = 0;
			__syncthreads();


			if ((thread_point >> 2) < xy_size)
			{
				const unsigned char avg_color = (pixels[thread_point] + pixels[thread_point + 1] + pixels[thread_point +
						2]) /
					3 / GLASS_MODE_COLOR_DIV; // calculate the avg color
				atomicAdd(&colors[avg_color], 1);


				__syncthreads();

				__shared__ unsigned char most_common_color;


				if (threadIdx.x == 0)
				{
					most_common_color = 0;
					auto count = -1;
					for (auto i = 0; i < GLASS_MODE_WARP_SIZE; i++)
					{
						if (count == -1 || colors[i] > count)
						{
							most_common_color = i;
							count = colors[i];
						}
					}
				}


				__syncthreads();

				int point_r = ((block_y / GLASS_MODE_WARP_SIZE_SQRT) * x_reduced + (block_x /
					GLASS_MODE_WARP_SIZE_SQRT));

				pixels_reduced[point_r] = most_common_color;
			}
			//pixels[thread_point] = most_common_color;
			//pixels[thread_point + 1] = most_common_color;
			//pixels[thread_point + 2] = most_common_color;

			// End
		}


		void reduce_noise(unsigned char* pixels_reduced, const int x_reduced, const int y_reduced,
		                  int xy_reduced)
		{
			//int colors[64] = { 0 };
			//for (int y = 0; y < y_reduced; y++)
			//	for (int x = 0; x < x_reduced; x++)
			//		colors[pixels_reduced[y * x_reduced + x]]++;


			const auto common_color_min_count = 50;
			const auto processed_pixel_flag = 128;

			//unsigned int xa_size = y_reduced * x_reduced;

			unsigned int xa_size = y_reduced * x_reduced;

			auto process = [&](int point, const int point_max, const int point_jump, const int max_count)
			{
				const auto point_start = point;
				auto point_end = point;
				const auto color = pixels_reduced[point];

				auto count = 0;
				for (; point < point_max; point += point_jump)
				{
					if (color == pixels_reduced[point])
					{
						point_end = point;
						count = 0;
					}
					else if (++count >= max_count)
					{
						break;
					}
				}

				if (point_start < point_end)
				{
					for (auto point_2 = point_start; point_2 <= point_end; point_2 += point_jump)
						pixels_reduced[point_2] = color;
				}


				return point_end;
			};


			for (auto y = 0; y < y_reduced - 1; y++)
			{
				auto point = y * x_reduced;
				const auto point_max = point + x_reduced - 1;
				while (point < point_max)
					point = process(point, point_max, 1, 5) + 1;
			}

			for (auto x = 0; x < x_reduced; x++)
			{
				auto point = x;
				const auto point_max = x + (y_reduced - 1) * x_reduced;
				while (point < point_max)
					point = process(point, point_max, x_reduced, 4) + x_reduced;
			}
		}


		__global__ void kernel_mark_shapes(unsigned char* pixels_reduced, const int x_reduced,
		                                   int y_reduced,
		                                   unsigned char* pixels,
		                                   const int x_size, int y_size,
		                                   const int xy_size,
		                                   const int x_end, const int y_end,
		                                   bool* image_area_data, const float texts_level,
		                                   const float background_level, const bool dark_background)
		{
			auto block_x = (blockIdx.x * GLASS_MODE_WARP_SIZE_SQRT) % x_end;
			block_x -= block_x % GLASS_MODE_WARP_SIZE_SQRT;
			const auto thread_x = threadIdx.x % GLASS_MODE_WARP_SIZE_SQRT;
			if (block_x + thread_x >= x_end) return;


			auto block_y = ((blockIdx.x * GLASS_MODE_WARP_SIZE_SQRT) / x_end) * GLASS_MODE_WARP_SIZE_SQRT;
			block_y -= block_y % GLASS_MODE_WARP_SIZE_SQRT;
			const auto thread_y = threadIdx.x / GLASS_MODE_WARP_SIZE_SQRT;
			if (block_y + thread_y >= y_end) return;


			if (image_area_data && image_area_data[(block_y + thread_y) * x_size + (block_x + thread_x)]) 
				return;


			const auto block_point = block_y * x_end + block_x;
			auto thread_point = block_point + thread_y * x_end + thread_x;

			const auto y_point = thread_point / x_end;
			const auto x_point = thread_point % x_end;

			thread_point *= 4;


			__shared__ int avg_shapes_avg_color;
			__shared__ int avg_shapes_count;
			if (threadIdx.x == 0)
			{
				avg_shapes_avg_color = 0;
				avg_shapes_count = 0;
			}

			__syncthreads();

			const auto reduced_point = (y_point / GLASS_MODE_WARP_SIZE_SQRT) * x_reduced + (x_point /
				GLASS_MODE_WARP_SIZE_SQRT);

			unsigned char avg_color = (((pixels[thread_point] + pixels[thread_point + 1] + pixels[thread_point + 2]) / 3
			) / GLASS_MODE_COLOR_DIV) * GLASS_MODE_COLOR_DIV;
			const unsigned char reduced_color = pixels_reduced[reduced_point] * GLASS_MODE_COLOR_DIV;

			const auto is_shape_color = avg_color != reduced_color;


			if (is_shape_color)
			{
				atomicAdd(&avg_shapes_count, 1);
				atomicAdd(&avg_shapes_avg_color, avg_color);
			}

			__syncthreads();

			if (threadIdx.x == 0)
			{
				avg_shapes_avg_color /= avg_shapes_count;
			}

			__syncthreads();


			if (is_shape_color)
			{
				if (avg_shapes_avg_color < reduced_color)
				{
					pixels[thread_point] = ~pixels[thread_point];
					pixels[thread_point + 1] = ~pixels[thread_point + 1];
					pixels[thread_point + 2] = ~pixels[thread_point + 2];

					avg_color = (((pixels[thread_point] + pixels[thread_point + 1] + pixels[thread_point + 2]) / 3) /
						GLASS_MODE_COLOR_DIV) * GLASS_MODE_COLOR_DIV;
				}
			}
			else
			{
				//pixels[thread_point] = pixels[thread_point + 1] = pixels[thread_point + 2] = pixels[thread_point + 3] = 0;

				if (dark_background)
				{
					if (reduced_color > 128)
					{
						pixels[thread_point] = ~pixels[thread_point];
						pixels[thread_point + 1] = ~pixels[thread_point + 1];
						pixels[thread_point + 2] = ~pixels[thread_point + 2];
					}
				}


				if (background_level != 0)
				{
					pixels[thread_point] *= background_level;
					pixels[thread_point + 1] *= background_level;
					pixels[thread_point + 2] *= background_level;
					pixels[thread_point + 3] = 255 * background_level;
				}
				else
				{
					memset(&pixels[thread_point], 0, sizeof(unsigned char) * 4);
					//pixels[thread_point] = pixels[thread_point + 1] = pixels[thread_point + 2] = pixels[thread_point + 3] = 0;
				}
			}

			__shared__ int shape_max_brightness;

			__syncthreads();
			if (threadIdx.x == 0)
			{
				shape_max_brightness = 0;
			}

			__syncthreads();

			if (is_shape_color)
			{
				atomicMax(&shape_max_brightness, avg_color);
			}

			__syncthreads();

			if (is_shape_color)
			{
				auto scalar = 255 / static_cast<float>(shape_max_brightness);

				avg_color *= scalar;
				//if (avg_color > 255) avg_color = 255;

				int b = pixels[thread_point];
				int g = pixels[thread_point + 1];
				int r = pixels[thread_point + 2];


				if (texts_level < 1.0)
				{
#if 0
					b *= texts_level;
					g *= texts_level;
					r *= texts_level;

#else
					scalar *= texts_level;
#endif
				}

				b *= scalar;
				g *= scalar;
				r *= scalar;

				if (b > 255) b = 255;
				if (g > 255) g = 255;
				if (r > 255) r = 255;


				pixels[thread_point] = b;
				pixels[thread_point + 1] = g;
				pixels[thread_point + 2] = r;
			}
		}


		bool map_shapes()
		{
			hipError_t cuda_result;
			if (images_level < 1.0)
			{
				kernel_perform_images_opacity
					<< < ((y_end) * (x_end) * 4) / GLASS_MODE_WARP_SIZE, GLASS_MODE_WARP_SIZE >> >
					(d_pixels, x_end, y_end, (y_end - 1) * x_end,
					 d_image_area_data, images_level);

				cuda_result = hipDeviceSynchronize();
				if (cuda_result != hipSuccess)
				{
					CudaCheckError(cuda_result);
					return false;
				}
			}


			kernel_build_reduced_pixels
				<< < xy_size_reduced, GLASS_MODE_WARP_SIZE >> >
				(d_pixels, x_end, (y_end - 1) * x_end, d_pixels_reduced, x_size_reduced);


			cuda_result = hipDeviceSynchronize();
			if (cuda_result != hipSuccess)
			{
				CudaCheckError(cuda_result);
				return false;
			}


			cuda_result = hipMemcpy(pixels_reduced, d_pixels_reduced, sizeof(unsigned char) * xy_size_reduced,
			                         hipMemcpyDeviceToHost);
			if (cuda_result != hipSuccess)
			{
				CudaCheckError(cuda_result);
				return false;
			}

			reduce_noise(pixels_reduced, x_size_reduced, y_size_reduced, xy_size_reduced);


#if 0 // Display reduced pixels (For debug only)
			for (int y = 0; y < data->y_end; y++)
				for (int x = 0; x < data->x_end; x++)
				{
					int point = (y * data->x_end + x) * 4;
					int point_r = ((y / GLASS_MODE_WARP_SIZE_SQRT) * data->x_size_reduced + (x / GLASS_MODE_WARP_SIZE_SQRT));

					pixels[point] = pixels[point + 1] = pixels[point + 2] = ((data->pixels_reduced[point_r]) & 63) * GLASS_MODE_COLOR_DIV;
				}

			return;
#endif

			cuda_result = hipMemcpy(d_pixels_reduced, pixels_reduced, sizeof(unsigned char) * xy_size_reduced,
			                         hipMemcpyHostToDevice);
			if (cuda_result != hipSuccess)
			{
				CudaCheckError(cuda_result);
				return false;
			}


			// GPU-Process(pixels_reduced,pixels)
			kernel_mark_shapes
				<< < xy_size_reduced, GLASS_MODE_WARP_SIZE >> >
				(d_pixels_reduced, x_size_reduced, y_size_reduced, d_pixels, x_size, y_size,
				 x_size * (y_size - 1), x_end, y_end,
				 d_image_area_data, shapes_level, background_level, dark_background);


			cuda_result = hipDeviceSynchronize();
			if (cuda_result != hipSuccess)
			{
				CudaCheckError(cuda_result);
				return false;
			}


			return true;
		}
	}

	void init(ID3D11DeviceContext* d3d_context)
	{
		process_layer_gpu::d3d_context = d3d_context;
	}

	bool enable_cache_buffer(const bool enable)
	{
		is_enable_cached_buffer = enable;
		if (!enable)
		{
			if (d_cached_pixels)
			{
				hipFree(d_cached_pixels);
				d_cached_pixels = nullptr;
			}

			if (d_is_new_pixels)
			{
				hipFree(d_is_new_pixels);
				d_is_new_pixels = nullptr;
			}
		}
		else
		{
			if (!d_is_new_pixels)
			{
				const auto result = hipMalloc(&d_is_new_pixels, sizeof(bool));
				if (result != hipSuccess)
				{
					CudaCheckError(result);
					return false;
				}
			}
		}

		return true;
	}

	bool init_frame(int x_size, int y_size, int x_end, int y_end)
	{
		process_layer_gpu::x_end = x_size;
		process_layer_gpu::y_end = y_size;
		process_layer_gpu::x_end = x_end ? x_end : x_size;
		process_layer_gpu::y_end = y_end ? y_end : y_size;
		return true;
	}

	void set_default_settings()
	{
		glass_effect::is_enabled = false;
		is_enable_cached_buffer = false;
	}

	void free_resources()
	{
		if (d_pixels)
		{
			hipFree(d_pixels);
			d_pixels = nullptr;
		}

		if (d_cached_pixels)
		{
			hipFree(d_cached_pixels);
			d_cached_pixels = nullptr;
		}

		if (d_image_area_data)
		{
			hipFree(d_image_area_data);
			d_image_area_data = nullptr;
		}

		if (cuda_resource)
		{
			hipGraphicsUnregisterResource(cuda_resource);
			cuda_resource = nullptr;
		}

		x_size = y_size = x_end = y_end = 0;

		EmptyWorkingSet(GetCurrentProcess()); // Reduce memory usage
	}

	bool begin_process(ID3D11Texture2D* texture, const int capture_x_size, const int capture_y_size)
	{
#if 0
		D3D11_MAPPED_SUBRESOURCE map_info;
		ZeroMemory(&map_info, sizeof(D3D11_MAPPED_SUBRESOURCE));

		const auto hr = d3d_context->Map
		(
			texture,
			0, // Subresource
			D3D11_MAP_READ,
			0, // MapFlags
			&map_info
		);

		if (hr != S_OK)
		{
			PLOGE << "Failed to get mapped cpu texture";
			return false;
		}

		process_layer_cpu::texture = texture;
		return load_frame(static_cast<byte*>(map_info.pData), map_info.RowPitch / 4,
			map_info.DepthPitch / map_info.RowPitch,
			x_end, y_end);
#endif

		hipError_t result;
		const auto is_resized = capture_x_size != x_end || capture_y_size != y_end;
		if (is_resized)
		{
			free_resources();

			x_end = capture_x_size;
			y_end = capture_y_size;


			D3D11_MAPPED_SUBRESOURCE map_info;
			ZeroMemory(&map_info, sizeof(D3D11_MAPPED_SUBRESOURCE));

			const auto hr = d3d_context->Map
			(
				texture,
				0, // Subresource
				D3D11_MAP_READ,
				0, // MapFlags
				&map_info
			);

			if (hr != S_OK)
			{
				PLOGE << "Failed to get mapped cpu texture";
				return false;
			}

			x_size = map_info.RowPitch / 4;
			y_size = map_info.DepthPitch / map_info.RowPitch;

			d3d_context->Unmap(texture, 0);


			result = cudaGraphicsD3D11RegisterResource(&cuda_resource, texture, hipGraphicsRegisterFlagsNone);
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				return false;
			}

			result = hipMalloc(&d_pixels, x_size * y_size * 4 * sizeof(unsigned char));
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				return false;
			}

			if (glass_effect::is_enabled)
				glass_effect::init();


			EmptyWorkingSet(GetCurrentProcess()); // Reduce memory usage
		}

		result = hipGraphicsMapResources(1, &cuda_resource, nullptr);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			return false;
		}


		result = hipGraphicsSubResourceGetMappedArray(&cu_array, cuda_resource, 0, 0);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			hipGraphicsUnmapResources(1, &cuda_resource, nullptr);
			return false;
		}

		result = hipMemcpyFromArray(d_pixels, cu_array, 0, 0, x_end * y_end * 4 * sizeof(unsigned char),
		                             hipMemcpyDeviceToDevice);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			hipGraphicsUnmapResources(1, &cuda_resource, nullptr);
			// TODO - Add unmap here
			return false;
		}


		if (is_enable_cached_buffer && is_resized)
		{
			if (d_cached_pixels)
			{
				hipFree(d_cached_pixels);
				d_cached_pixels = nullptr;
			}

			result = hipMalloc(&d_cached_pixels, x_size * y_size * 4 * sizeof(unsigned char));
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				return false;
			}

			result = hipMemcpy(d_cached_pixels, d_pixels, x_size * y_size * 4 * sizeof(unsigned char),
			                    hipMemcpyDeviceToDevice);
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				return false;
			}
		}


		return true;
	}

	bool end_process()
	{
		auto result = hipMemcpyToArray(cu_array, 0, 0, d_pixels, x_end * y_end * 4 * sizeof(unsigned char),
		                                hipMemcpyDeviceToDevice);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			return false;
		}

		result = hipGraphicsUnmapResources(1, &cuda_resource, nullptr);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			return false;
		}


		return true;
	}


	__global__ void kernel_is_new_pixels(bool* image_area_data, unsigned char* cached_pixels, unsigned char* pixels,
	                                     const int x_size, const int x_end, const int y_end, bool* is_new_pixels)
	{
		auto point = blockIdx.x * blockDim.x + threadIdx.x;
		if (point % x_size >= x_end)
			return;

		if (point / x_size >= y_end)
			return;

		point *= 4;

		auto _is_new_pixels = false;
		if (cached_pixels[point] != pixels[point])
		{
			cached_pixels[point] = pixels[point];
			_is_new_pixels = true;
		}

		if (cached_pixels[point + 1] != pixels[point + 1])
		{
			cached_pixels[point + 1] = pixels[point + 1];
			_is_new_pixels = true;
		}

		if (cached_pixels[point + 2] != pixels[point + 2])
		{
			cached_pixels[point + 2] = pixels[point + 2];
			_is_new_pixels = true;
		}

		if (_is_new_pixels && !(*is_new_pixels))
			*is_new_pixels = true;
	}

	bool is_new_pixels(bool& error)
	{
		auto result = hipMemset(d_is_new_pixels, false, sizeof(bool));
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}

		kernel_is_new_pixels
			<< < (x_size * y_size) / DARK_MODE_WARP_SIZE, DARK_MODE_WARP_SIZE >> >
			(d_image_area_data, d_cached_pixels, d_pixels, x_size, x_end, y_end, d_is_new_pixels);

		result = hipDeviceSynchronize();
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}

		bool is_new_pixels = false;
		result = hipMemcpy(&is_new_pixels, d_is_new_pixels, sizeof(bool), hipMemcpyDeviceToHost);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}

		return is_new_pixels;
	}

	bool set_image_area_data(bool* image_area_data)
	{
		if (!image_area_data)
		{
			if (d_image_area_data)
			{
				hipFree(d_image_area_data);
				d_image_area_data = nullptr;
			}
			return true;
		}

		if (!d_image_area_data)
		{
			const auto result = hipMalloc(&d_image_area_data, x_size * y_size * sizeof(bool));
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				return false;
			}
		}

		const auto result = hipMemcpy(d_image_area_data, image_area_data, x_size * y_size * sizeof(bool),
		                               hipMemcpyHostToDevice);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			return false;
		}

		return true;
	}

	__global__ void kernel_is_current_pixels_bright(unsigned char* pixels, bool* image_area_data, int* bright_count,
	                                                const int x_size, const int x_end, const int y_end)
	{
		auto point = blockIdx.x * blockDim.x + +threadIdx.x;;
		if (point % x_size >= x_end)
			return;

		if (point / x_size >= y_end)
			return;


		point *= 4;

		const auto color_avg = (pixels[point] + pixels[point + 1] + pixels[point + 2]) / 3;
		if (color_avg > 127)
			atomicAdd(bright_count, 1);
	}

	bool is_current_pixels_bright(bool& error)
	{
		if (!d_bright_pixels_count)
		{
			const auto result = hipMalloc(&d_bright_pixels_count, sizeof(int));
			if (result != hipSuccess)
			{
				CudaCheckError(result);
				error = true;
				return false;
			}
		}

		auto result = hipMemset(d_bright_pixels_count, 0, sizeof(int));
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}


		kernel_is_current_pixels_bright
			<< < (x_size * y_size) / DARK_MODE_WARP_SIZE, DARK_MODE_WARP_SIZE >> >
			(d_pixels, d_image_area_data, d_bright_pixels_count, x_size, x_end, y_end);


		result = hipDeviceSynchronize();
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}


		int bright_pixels_count = 0;
		result = hipMemcpy(&bright_pixels_count, d_bright_pixels_count, sizeof(int), hipMemcpyDeviceToHost);
		if (result != hipSuccess)
		{
			CudaCheckError(result);
			error = true;
			return false;
		}


		return bright_pixels_count > x_end * y_end / 2;
	}


	__global__ void kernel_invert_colors
	(bool* d_image_area_data, unsigned char* d_pixels, const int x_size, const int x_end, const int y_end)
	{
		const auto idx = blockIdx.x * blockDim.x + threadIdx.x;


		const auto x = idx % x_size;
		const auto y = idx / x_size;

		if (x >= x_end || y >= y_end)
			return;


		if (d_image_area_data && d_image_area_data[idx])
			return;

		const auto point = (y * x_end + x) * 4;

		d_pixels[point] = ~d_pixels[point];
		d_pixels[point + 1] = ~d_pixels[point + 1];
		d_pixels[point + 2] = ~d_pixels[point + 2];
	}

	bool invert_colors()
	{
		//return true;
		kernel_invert_colors
			<< < (x_size * y_size) / DARK_MODE_WARP_SIZE, DARK_MODE_WARP_SIZE >> >
			(d_image_area_data, d_pixels, x_size, x_end, y_end);


		const auto result = hipDeviceSynchronize();
		if (result == hipSuccess)
			return true;

		CudaCheckError(result);
		return false;
	}
}
